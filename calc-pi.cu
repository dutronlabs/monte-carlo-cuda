#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

#define THREADS_PER_BLOCK 256

__global__ void monte_carlo_pi(int iterations, hiprandState *state, unsigned int *result)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int points_in_circle = 0;

    hiprand_init(1234, tid, 0, &state[tid]);

    for (int i = 0; i < iterations; i++) {
        float x = hiprand_uniform(&state[tid]);
        float y = hiprand_uniform(&state[tid]);

        if (x * x + y * y <= 1.0f) {
            points_in_circle++;
        }
    }

    result[tid] = points_in_circle;
}

int main() {
    monte_carlo_pi(1000000, 1, 1)
}